#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace HYSLAM { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
